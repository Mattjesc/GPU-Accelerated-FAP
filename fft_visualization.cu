#include "hip/hip_runtime.h"
#include "fft_visualization.h"
#include <hipfft/hipfft.h>
#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <vector>
#include <complex>
#include <algorithm> // Include this header for std::max_element

// Error handling macro
#define CHECK(call)                                                        \
{                                                                          \
    const hipError_t error = call;                                        \
    if (error != hipSuccess)                                              \
    {                                                                      \
        std::cerr << "Error: " << __FILE__ << ":" << __LINE__ << ", "      \
                  << hipGetErrorString(error) << std::endl;               \
        exit(1);                                                           \
    }                                                                      \
}

// Function to calculate the magnitude of complex numbers
void calculateMagnitude(const std::vector<hipfftComplex>& data, std::vector<float>& magnitude) {
    for (size_t i = 0; i < data.size(); i++) {
        magnitude[i] = sqrt(data[i].x * data[i].x + data[i].y * data[i].y);
    }
}

// Function to calculate the phase of complex numbers
void calculatePhase(const std::vector<hipfftComplex>& data, std::vector<float>& phase) {
    for (size_t i = 0; i < data.size(); i++) {
        phase[i] = atan2(data[i].y, data[i].x);
    }
}

// Function to generate input data
void generateInputData(std::vector<hipfftComplex>& h_data, int N, int signalType) {
    h_data.resize(N);
    for (int i = 0; i < N; i++) {
        float t = 2 * M_PI * i / N;
        switch (signalType) {
            case 1: // Sine wave
                h_data[i].x = sin(t);
                h_data[i].y = 0.0f;
                break;
            case 2: // Square wave
                h_data[i].x = (i % (N / 2) < (N / 4)) ? 1.0f : -1.0f;
                h_data[i].y = 0.0f;
                break;
            case 3: // Sawtooth wave
                h_data[i].x = 2 * (t / (2 * M_PI) - floor(0.5 + t / (2 * M_PI)));
                h_data[i].y = 0.0f;
                break;
            case 4: // Triangle wave
                h_data[i].x = 2 * abs(2 * (t / (2 * M_PI) - floor(0.5 + t / (2 * M_PI)))) - 1;
                h_data[i].y = 0.0f;
                break;
            default:
                std::cerr << "Unknown signal type" << std::endl;
                exit(1);
        }
    }
}

// FFT function
void fftOnDevice(std::vector<hipfftComplex>& h_data) {
    hipfftHandle plan;
    hipfftComplex *d_data;
    size_t size = sizeof(hipfftComplex) * h_data.size();

    // Allocate device memory
    CHECK(hipMalloc(&d_data, size));
    CHECK(hipMemcpy(d_data, h_data.data(), size, hipMemcpyHostToDevice));

    // Create FFT plan
    if (hipfftPlan1d(&plan, h_data.size(), HIPFFT_C2C, 1) != HIPFFT_SUCCESS) {
        std::cerr << "CUFFT error: Plan creation failed" << std::endl;
        hipFree(d_data);
        exit(1);
    } else {
        std::cout << "CUFFT Plan creation successful." << std::endl;
    }

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    // Record start event
    CHECK(hipEventRecord(start, 0));

    // Execute FFT
    if (hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        std::cerr << "CUFFT error: ExecC2C Forward failed" << std::endl;
        hipfftDestroy(plan);
        hipFree(d_data);
        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
        exit(1);
    } else {
        std::cout << "CUFFT execution successful." << std::endl;
    }

    // Synchronize to ensure FFT computation is complete
    CHECK(hipDeviceSynchronize());

    // Record stop event
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    // Compute elapsed time
    float elapsed_time;
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    std::cout << "FFT computation time: " << elapsed_time << " ms" << std::endl;

    // Copy result back to host
    CHECK(hipMemcpy(h_data.data(), d_data, size, hipMemcpyDeviceToHost));

    // Clean up
    hipfftDestroy(plan);
    hipFree(d_data);
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
}

void performFFTAnalysis(int N, int signalType, float scaleFactor, float minFrequency, float maxFrequency, bool displayPhase) {
    std::vector<hipfftComplex> h_data;
    generateInputData(h_data, N, signalType);

    // Debug: Print input data
    std::cout << "Input Data:" << std::endl;
    for (int i = 0; i < N; i++) {
        std::cout << "Data[" << i << "]: (" << h_data[i].x << ", " << h_data[i].y << ")" << std::endl;
    }

    // Perform FFT on Device
    fftOnDevice(h_data);

    // Calculate magnitude spectrum
    std::vector<float> magnitude(N);
    calculateMagnitude(h_data, magnitude);

    // Calculate phase spectrum
    std::vector<float> phase(N);
    calculatePhase(h_data, phase);

    // Normalize magnitude spectrum
    float maxMagnitude = *std::max_element(magnitude.begin(), magnitude.end());
    for (auto& mag : magnitude) {
        mag /= maxMagnitude;
        mag *= scaleFactor;
    }

    // Output magnitude spectrum within the specified frequency range
    std::cout << "Magnitude Spectrum:" << std::endl;
    for (int i = 0; i < N; i++) {
        float frequency = static_cast<float>(i) / N * 2 * M_PI;
        if (frequency >= minFrequency && frequency <= maxFrequency) {
            std::cout << "Magnitude[" << i << "]: " << magnitude[i] << std::endl;
        }
    }

    // Output phase spectrum within the specified frequency range if requested
    if (displayPhase) {
        std::cout << "Phase Spectrum:" << std::endl;
        for (int i = 0; i < N; i++) {
            float frequency = static_cast<float>(i) / N * 2 * M_PI;
            if (frequency >= minFrequency && frequency <= maxFrequency) {
                std::cout << "Phase[" << i << "]: " << phase[i] << std::endl;
            }
        }
    }

    // Output frequency spectrum within the specified frequency range
    std::cout << "Frequency Spectrum:" << std::endl;
    for (int i = 0; i < N; i++) {
        float frequency = static_cast<float>(i) / N * 2 * M_PI;
        if (frequency >= minFrequency && frequency <= maxFrequency) {
            std::cout << "Frequency[" << i << "]: " << frequency << std::endl;
        }
    }
}